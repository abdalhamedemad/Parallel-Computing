#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define MAX_NODES 1000
#include <iostream>
#include <fstream>
#include <vector>
#include "utils1.h"
using namespace std;

__global__
void edge_centric(unsigned int *d_row, unsigned int *d_col, unsigned int *d_row_size, unsigned int *d_col_size, unsigned int *d_level, unsigned int *d_new_vertex_visited, unsigned int *d_current_level, unsigned int *d_num_nodes) {
    unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;
    if (edge < *d_col_size) {
        unsigned int vertex = d_row[edge];
        unsigned int neighbor = d_col[edge];
        if (d_level[vertex] == *d_current_level - 1 && d_level[neighbor] == UINT_MAX) {
            d_level[neighbor] = *d_current_level;
            *d_new_vertex_visited = 1;
        }
    }
}

int main(int argc, char *argv[]) {
    string fileName = argv[1];
    unsigned int num_nodes;
    unsigned int num_edges;

    vector<vector<unsigned int>> adjacency_list = read_adjacency_list(fileName, num_nodes,num_edges);

    COO coo;
    convert_adj_list_to_coo(adjacency_list, num_nodes, coo);

    vector<unsigned int> level(num_nodes, UINT_MAX);
    level[0] = 0;
    unsigned int new_vertex_visited = 0;
    unsigned int current_level = 0;

    unsigned int *d_row, *d_col;
    unsigned int *d_row_size, *d_col_size;
    unsigned int *d_level;
    unsigned int *d_new_vertex_visited;
    unsigned int *d_current_level;
    unsigned int *d_num_nodes;

    hipMalloc(&d_row, coo.size * sizeof(unsigned int));
    hipMalloc(&d_col, coo.size * sizeof(unsigned int));
    hipMalloc(&d_row_size, sizeof(unsigned int));
    hipMalloc(&d_col_size, sizeof(unsigned int));
    hipMalloc(&d_level, num_nodes * sizeof(unsigned int));
    hipMalloc(&d_new_vertex_visited, sizeof(unsigned int));
    hipMalloc(&d_current_level, sizeof(unsigned int));
    hipMalloc(&d_num_nodes, sizeof(unsigned int));

    hipMemcpy(d_row, coo.row.data(), coo.size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, coo.col.data(), coo.size * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_size, &coo.size, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_size, &coo.size, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_level, level.data(), num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_num_nodes, &num_nodes, sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int numOfThreads = 128;
    unsigned int numOfBlocks = (coo.size + numOfThreads - 1) / numOfThreads;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    new_vertex_visited = 1;
    for (unsigned int current_level = 1; new_vertex_visited; ++current_level) {
        new_vertex_visited = 0;
        hipMemcpyAsync(d_current_level, &current_level, sizeof(unsigned int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(d_new_vertex_visited, &new_vertex_visited, sizeof(unsigned int), hipMemcpyHostToDevice, stream1);

        edge_centric<<<numOfBlocks, numOfThreads, 0, stream2>>>(d_row, d_col, d_row_size, d_col_size, d_level, d_new_vertex_visited, d_current_level, d_num_nodes);

        hipMemcpyAsync(&new_vertex_visited, d_new_vertex_visited, sizeof(unsigned int), hipMemcpyDeviceToHost, stream2);
        hipStreamSynchronize(stream2);
    }

    hipMemcpy(level.data(), d_level, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);

    ofstream output_file;
    output_file.open("output.txt");
    for (int i = 0; i < num_nodes; ++i) {
        output_file << level[i] << endl;
    }

    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_row_size);
    hipFree(d_col_size);
    hipFree(d_level);
    hipFree(d_new_vertex_visited);
    hipFree(d_current_level);
    hipFree(d_num_nodes);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
