#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

// kernel3: each thread produces one output matrix column
__global__ void matrixAdd3(float* A, float* B, float* C, int rows, int cols) {
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (j < cols) {
		for (int i = 0; i < rows; i++) {
			C[i * cols + j] = A[i * cols + j] + B[i * cols + j];
		}
	}
}
// fix kernel 3 to work on size of 1*1000
__global__ void matrixAdd3_1_1000(float* A, float* B, float* C, int rows, int cols) {
	int j = threadIdx.y + blockIdx.y * blockDim.x;
	if (j < cols) {
		for (int i = 0; i < rows; i++) {
			C[i * cols + j] = A[i * cols + j] + B[i * cols + j];
		}
	}
}
// fix kernel 3 to work on size of 1000*1




int main(int argc, char *argv[]) {
	printf("Reading file...\n");

	const char* filename = argv[1];
  const char* outputFilename = argv[2];
	int numOfTests, rows, cols;
	
	FILE* file = fopen(filename, "r");
	FILE* outputFile = fopen(outputFilename, "w");
	
	if (file == NULL) {
		printf("Error: can't open file.\n");
		exit(1);
	}
	fscanf(file, "%d", &numOfTests);
	
	for (int i = 0; i < numOfTests; i++) {
		
		fscanf(file, "%d", &rows);
		fscanf(file, "%d", &cols);
		
		float* A = (float*)malloc(rows * cols * sizeof(float));
		float* B = (float*)malloc(rows * cols * sizeof(float));
		
		for (int i = 0; i < rows; i++) {
			for (int j = 0; j < cols; j++) {
				fscanf(file, "%f", &A[i * cols + j]);
			}
		}
		for (int i = 0; i < rows; i++) {
			for (int j = 0; j < cols; j++) {
				fscanf(file, "%f", &B[i * cols + j]);
			}
		}
		// Allocate memory on the device
		float* d_A, * d_B, * d_C;
		hipMalloc(&d_A, rows * cols * sizeof(float));
		hipMalloc(&d_B, rows * cols * sizeof(float));
		hipMalloc(&d_C, rows * cols * sizeof(float));

		// Copy data to the device
		hipMemcpy(d_A, A, rows * cols * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B, rows * cols * sizeof(float), hipMemcpyHostToDevice);

		dim3 threadsPerBlock(32, 32);

		dim3 blocksPerGrid((int)ceil(float(rows) / threadsPerBlock.x), (int)ceil(float(cols) / threadsPerBlock.y));
		// Copy data back to host
		float* C = (float*)malloc(rows * cols * sizeof(float));
		// print matrix C with kernel 3
		matrixAdd3 << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, rows, cols);
		hipMemcpy(C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
		
		printf("Writing to file...\n");
		if (outputFile == NULL) {
			printf("Error: can't open file.\n");
			exit(1);
		}
		for (int i = 0; i < rows; i++) {
			for (int j = 0; j < cols; j++) {
				fprintf(outputFile, "%.1f ", C[i * cols + j]);
				//fprintf(outputFile, "%f ", C[i * cols + j]);
			}
			fprintf(outputFile, "\n");
		}
	printf("Done writing to file.\n");
		// Free memory
		free(A);
		free(B);
		free(C);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}
	fclose(file);
	fclose(outputFile);

	return 0;
}
